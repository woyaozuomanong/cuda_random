#include "hip/hip_runtime.h"
#include "hip/hip_runtime.h"
#include "" 
#include "hiprand/hiprand_kernel.h" 

#include <stdio.h>
#include <sys/time.h>


double cpuSecond()
{
#if _WIN32
	_LARGE_INTEGER time_start;    /*开始时间*/
	double dqFreq;                /*计时器频率*/
	LARGE_INTEGER f;            /*计时器频率*/
	QueryPerformanceFrequency(&f);
	dqFreq = (double)f.QuadPart;
	QueryPerformanceCounter(&time_start);
	return time_start.QuadPart / dqFreq ;//单位为秒，精度为1000 000/（cpu主频）微秒
#endif

#if __linux__
	struct timeval tp;
	gettimeofday(&tp, NULL);
	return ((double)tp.tv_sec + (double)tp.tv_usec*1e-6);
#endif
}



__global__ void kernel_random(float *dev_random_array,int height,int width,long clock_for_rand)
{
    int x = threadIdx.x + blockIdx.x*blockDim.x;
    int y = threadIdx.y + blockIdx.y*blockDim.y;

    if(x<0 || x>width || y<0 || y>height)
    {
        return;
    }

    int pos = y*width + x;

    hiprandState state;
    hiprand_init(pos,pos,0,&state);
    dev_random_array[pos] = abs(hiprand_uniform(&state));
}


int main()
{
    double iStart,iElapse;
    iStart=cpuSecond();
    const int array_size_width = 10;
    const int array_size_height = 10;
    float random_array[array_size_width*array_size_height];
    for(int i=0;i<array_size_width*array_size_height;i++)
    {
        random_array[i] = 0;
    }

    //error status
    hipError_t cuda_status;

    //only chose one GPU
    cuda_status = hipSetDevice(0);
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"hipSetDevice failed! Do you have a CUDA-Capable GPU installed?");
        return 0;
    }

    float *dev_random_array;

     //allocate memory on the GPU
    cuda_status = hipMalloc((void**)&dev_random_array,sizeof(float)*array_size_width*array_size_height);
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"dev_reference_image hipMalloc Failed");
        exit( EXIT_FAILURE );
    }


    dim3 threads(16,16);
    dim3 grid(max(array_size_width/threads.x,1),max(array_size_height/threads.y,1));

    long clock_for_rand = clock();
    printf("clock=%d\n",clock_for_rand);
    kernel_random<<<grid,threads>>>(dev_random_array,array_size_width,array_size_height,clock_for_rand);

    //copy out the result
    cuda_status = hipMemcpy(random_array,dev_random_array,sizeof(float)*array_size_width*array_size_height,hipMemcpyDeviceToHost);//dev_depthMap
    if(cuda_status != hipSuccess)
    {
        fprintf(stderr,"hipMemcpy Failed");
        exit( EXIT_FAILURE );
    }

     for(int i=0;i<array_size_width*array_size_height;i++)
     {
         printf("%f\n",random_array[i]);
     }

    iElapse=cpuSecond()-iStart;
    printf("Total time: %f\n",iElapse);
    //free
    hipFree(dev_random_array);
    return 0;
}
